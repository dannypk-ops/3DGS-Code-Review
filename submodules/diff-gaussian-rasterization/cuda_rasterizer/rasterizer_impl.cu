#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	// 전체 grid에서 현재 thread의 idx 반환
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];	// 현재 thread가 담당하는 Gaussian의 offset(영향을 준 tile의 갯수)를 저장
		uint2 rect_min, rect_max;

		// // 해당 point의 중심으로부터 max_radius만큼의 사각형을 생성하여, touch하는 grid의 index를 생성 ( rect_min, rect_max )
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;	// touch한 grid의 index
				key <<= 32;		// key(tile ID)를 32bit(4byte = INT)만큼 left shift 연산
				key |= *((uint32_t*)&depths[idx]);	// depth의 정보를 OR 연산을 통해 key에 입력
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
				// 누적합으로 생성하였던 offset은 다음과 같이 key : value의 index를 표현하기 위해 이용!
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	// 현재 thread의 index를 가리킨다.
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;		// right shift연산, tile grid 정보값만이 남게 된다.

	// 만약 첫 Gaussian(idx == 0)이거나 또는 마지막 Gaussian(idx = L-1)인 경우에는 위치가 고정 ( 정렬된 결과이기 때문 )
	// 첫번째 Gaussian은 첫번째 grid의 시작 Gaussian
	// 마지막 Gaussian은 마지막 grid의 마지막 Gaussian
	if (idx == 0)
		ranges[currtile].x = 0;
	// 중간 Gaussian들에 대해서는 이전 Gaussian과의 관계를 통해서 결정해야 한다.
	// 바로 이전의 Gaussian이 속하는 tile의 정보와 다르다면, 각 Gaussian이 tile의 경계를 가리키는 Gaussian이기 때문에
	// 다음과 같이 값을 설정해 줄 수 있다.
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// GeometryState 구조체의 멤벼 변수들에 대해서 필요한 메모리를 할당해 주는 함수
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	/*
	template <typename T>
	static void obtain(char*& chunk, T*& ptr, std::size_t count, std::size_t alignment)
	{
		std::size_t offset = (reinterpret_cast<std::uintptr_t>(chunk) + alignment - 1) & ~(alignment - 1);
		ptr = reinterpret_cast<T*>(offset);re
		chunk = reinterpret_cast<char*>(ptr + count);
	}
	*/
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	// InclusiveSum에 필요한 임시 버퍼 ( geom.scan_size의 크기 )를 계산하기 위해서 호출
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* depth,
	bool antialiasing,
	int* radii,
	bool debug)
{
	// Fov로부터 focal length를 연산. ( intrinsic )
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	// GeometryState는 rasterizer_impl.h에 정의되어 있는 구조체
	// 각 Gaussians들의 정보들이 정의
	size_t chunk_size = required<GeometryState>(P);	// GeomteryState의 size를 연산
	char* chunkptr = geometryBuffer(chunk_size);	// size를 바탕으로, 외부에서 정의한 GeometryBuffer의 크기를 수정해 주는 함수 호출
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);	// point의 갯수만큼 chunk를 갖는 Buffer 생성 ( chunkptr 메모리 할당 )

	if (radii == nullptr)	// False
	{
		radii = geomState.internal_radii;
	}
	
	// config.h에 BLOCK_X , BLOCK_Y = 16으로 정의
	// width, height의 정보를 이용해서 (16 * 16) tile의 인덱스를 생성
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);	// geomstate와 동일하게 메모리 할당
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	// render하기에 앞서, 각 Gaussian들의 정보들을 preprocessing 하는 함수
	// 각 Gaussian의 2D Covariance, radii, color, tile_touched 등을 결정하는 함수이다.
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered,
		antialiasing
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	// 각 Gaussian이 결과 배열에서 데이터를 사용할 시작 위치(offset)을 구하는데 이용이 된다.
	// geomState.point_offsets에 누적합의 결과가 반환된다.
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	// num_rendered = 전체 Gaussian들이 영향을 미치는 tile의 총 갯수를 의미한다.
	// 누적합이기 때문에, 가장 마지막 List의 값이 전체 tile의 총 갯수를 의미하게 된다.
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);	// BinningState가 num_rendered개 데이터를 처리할 때 필요한 메모리 크기를 계산
	char* binning_chunkptr = binningBuffer(binning_chunk_size);		// binning_chunk_size만큼 GPU 메모리를 할당하고 시작 주소를 반환
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);	// 할당된 메모리에서 BinningState 멤버들을 num_rendered개에 맞게 초기화

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	// paper에 나왔던 그대로 { [ tile | depth ] : Gaussian ID } 형태의 dictionary를 생성
	// ( binningState.point_list_keys_unsorted, binningState.point_list_unsorte에 저장 )
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	// getHigherMsb = CPU에서 32비트 정수 중 가장 높은 1비트의 위치를 찾는 helper 함수이다.
	// 즉 이는 tile ID를 표현하는데 필요한 최소 비트 수를 의미한다.
	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	// 결국 정렬된 list들은 tileID 별로 우선 정렬이 된 이후에
	// depth가 낮은것부터 정렬되어 
	// 결과적으로는 tile별 depth 정보가 담겨있는 list가 반환되게 된다. ( 오름차순 )
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)	// 32 + bit는 RadixSort에서 정렬할 키의 총 비트 수를 의미한다. ( for memory efficiency )

	// tile_grid.x * tile_grid_y * sizeof(uint2) 크기 만큼의 memory를 0으로 초기화하여 메모리 할당
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	// 각 tile 별로 시작 Gaussian의 idx와 마지막 Gaussian idx의 정보를 담는 자료구조를 생성 ( imgState.ranges )
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	// geomState.rgb는 preprocess 과정에서 SH를 RGB의 형태로 변환하여 생성이 되는 값이다.
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		depth), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	const float* dL_invdepths,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dinvdepth,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool antialiasing,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		geomState.depths,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_invdepths,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor,
		dL_dinvdepth), debug);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		opacities,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		dL_dinvdepth,
		dL_dopacity,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		antialiasing), debug);
}
