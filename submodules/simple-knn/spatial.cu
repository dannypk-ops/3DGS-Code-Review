#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);

  // means에는 각 point의 최근접 3개의 point 사이의 distance의 평균이 저장된다.
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}